#include "hip/hip_runtime.h"
//Author: Adriel Kim
//6-27-2020
//Updated 7-5-2020
/*
Desc: Basic 2D matrix operations - element-wise addition, subtraction, multiplication, and division.

To do:
- Use vector instead of array?
- Be able to test for varying sizes of images. (For now we manually define with constant N)
- Add timer to compare CPU and GPU implementations
- Double check if all memory is freed
- Optimize by eliminating redundant calculations
- Test code on department servers
*/

#include "hip/hip_runtime.h"
#include ""
#include <sys/time.h>
#include <iostream>
#include <fstream>
#include <cstring>
#include <string>
#include <stdlib.h>
#include <stdio.h>

//define imin(a,b)  (a<b?a:b)//example of ternary operator in c++
//4176,2048
#define R 4176
#define C 2048
#define N (R*C)//# of elements in matrices
const int threadsPerBlock = 1024;//threads in a block. A chunk that shares the same shared memory.
const int blocksPerGrid = 8352;//imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);//this will be our output array size for sumKernel.

using namespace std;

hipError_t matrixOperation(double* c, const double* a, const double* b, unsigned int arrSize, int operation, float* kernel_runtime, float* GPU_transfer_time);
void CPUMatrixOperation(double* c, const double* a, const double* b, unsigned int arrSize, int operation);
long long start_timer();
long long stop_timer(long long start_time, const char *name);

//any advantages with mapping directly to strucutre of matrix? We're just representing 2D matrix using 1D array...
//it would be difficult to do the above since we want the operations to occur over abitrarily large matrices
//this can definitely be optimzied by elminating redundant calculations
__global__ void matrixAddKernel(double *c, const double *a, const double *b) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        //adds total number of running threads to tid, the current index.
        tid += blockDim.x * gridDim.x;
    }
}
__global__ void matrixSubtractKernel(double* c, const double* a, const double* b) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] - b[tid];
        //adds total number of running threads to tid, the current index.
        tid += blockDim.x * gridDim.x;
    }
}
__global__ void matrixMultiplyKernel(double* c, const double* a, const double* b) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

__global__ void matrixDivideKernel(double* c, const double* a, const double* b) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = (a[tid]/b[tid]);
        tid += blockDim.x * gridDim.x;
    }
}
//---------------------------------------------------------------------------------
void CPUMatrixAdd(double* c, const double* a, const double* b, unsigned int arrSize){
    for(int i = 0; i < arrSize; i++){
        c[i] = a[i] + b[i];
    }
}
void CPUMatrixSubtract(double* c, const double* a, const double* b, unsigned int arrSize){
    for(int i = 0; i < arrSize; i++){
        c[i] = a[i] - b[i];
    }
}

void CPUMatrixMultiply(double* c, const double* a, const double* b, unsigned int arrSize){
    for(int i = 0; i < arrSize; i++){
        c[i] = a[i] * b[i];
    }
}

void CPUMatrixDivide(double* c, const double* a, const double* b, unsigned int arrSize){
    for(int i = 0; i < arrSize; i++){
        c[i] = a[i] / b[i];
    }
}
//----------------------------------------------------------------------------------
void printMatrix(double* arr) {
    for (int i = 0;i < R; i++) {
        for (int k = 0;k < C; k++) {
            cout << (arr[k + i * R])<<" ";
        }
        cout << endl;
    }
}

void fillWithFitImage(string fileName, double* arr){
    string line;
    ifstream fitsFile;
    fitsFile.open("./FitTextFiles/"+fileName);
    int ind = 0;
    if(fitsFile.is_open()){
        getline(fitsFile, line);//gets rid of initial part, which is just shape
    }

    if(fitsFile.is_open()){
        while(getline(fitsFile, line) && ind < N){
            char cstr[line.size()+1];
            strcpy(cstr, line.c_str());
            double num = atof(cstr);
            //add num to arr
            arr[ind] = num;
            ind++;
        }
        fitsFile.close();
    }
    else{
        cout<<"Error opening file"<<endl;
    }

}

bool checkEquality(double* arr1, double* arr2, int arrSize){
    for(int i = 0;i < arrSize;i++){
        if (arr1[i]!=arr2[i]){
            return false;
        }
    }
    return true;

}

int main()
{
    //const int rows = R;
    //const int cols = C;
    //int inc = 0;

    //Must allocate host memory first before calling kernel.
    double* outputs = (double*)malloc(N * sizeof(double));
    double* doubleMatrix = (double*)malloc(N * sizeof(double));
    double* doubleMatrix2 = (double*)malloc(N * sizeof(double));
    double* CPUoutputs = (double*)malloc(N * sizeof(double));

    int operation = 0;
    
    cout << "Enter which operation (1 = add, 2 = subtract, 3 = multiply, 4 = divide)" << endl;
    cin >> operation;
    //populated 2D array with data
    cout<<"Populating image data"<<endl;
    fillWithFitImage("imgraw1.txt", doubleMatrix);
    fillWithFitImage("img1.txt", doubleMatrix2);

    cout<<"GPU Start!"<<endl;

    float* GPU_kernel_time = (float*)malloc(sizeof(float));
    float* GPU_transfer_time = (float*)malloc(sizeof(float));
    long long gpu_start_time = start_timer();
    hipError_t cudaStatus = matrixOperation(outputs, doubleMatrix, doubleMatrix2, N, operation,GPU_kernel_time, GPU_transfer_time);
    long long totalGPUTime = stop_timer(gpu_start_time, "Total GPU time");//is slowness in the kernel? find out. get kerne ltime
    
    cout << "GPU DONE!" << endl;
    cout<<"Total GPU Time: "<<totalGPUTime<<" microseconds"<<endl;
    cout<<"Kernel GPU Time: "<<*GPU_kernel_time<<" microseconds"<<endl;
    cout<<"Kernel Memory transfer time:" <<*GPU_transfer_time<<" microseconds"<<endl;
    //printMatrix(outputs);

    

    cout << "CPU Start!" << endl;

    long long cpu_start_time = start_timer();
    CPUMatrixOperation(CPUoutputs, doubleMatrix, doubleMatrix2, N, operation);
    long long totalCPUTime = stop_timer(cpu_start_time, "Total CPU time");

    cout << "CPU DONE!" << endl;
    cout<<"Total CPU Time: "<<totalCPUTime<<" microseconds"<<endl;

    //printMatrix(CPUoutputs);
    bool equal = checkEquality(outputs, CPUoutputs, N);
    if(equal == true)
        cout<<"CPU and GPU outputs are equal"<<endl;
    else
        cout<<"CPU and GPU outputs are NOT equal"<<endl;

    float ratio;
    ratio = (float)(totalCPUTime/(*GPU_kernel_time));
    cout<<"Speed up based on GPU Kernel Time: "<<ratio<<endl;

    free(outputs);
    free(doubleMatrix);
    free(doubleMatrix2);
    free(CPUoutputs);
    free(GPU_kernel_time);

    return 0;
}
hipError_t matrixOperation(double* c, const double* a, const double* b, unsigned int arrSize, int operation, float* kernel_runtime, float* GPU_transfer_time) {
    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    float kernel_time = 0;
    float transfer_time = 0;
    hipError_t cudaStatus;


    long long GPU_memory;
    GPU_memory = start_timer();

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    //Allocate GPU buffers for three vectors (two input, one output)
    cudaStatus = hipMalloc((void**)&dev_c, N * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 1");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, N * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 2");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, N * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 3");
        goto Error;
    }

    //Copy input vectors from host memory to GPU buffers
    cudaStatus = hipMemcpy(dev_a, a, sizeof(double) * N, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 1");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, sizeof(double) * N, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 2");
        goto Error;
    }
    transfer_time += stop_timer(GPU_memory, "GPU memory transfer time:");
    *GPU_transfer_time = transfer_time;

    long long kernel_start_time;
    kernel_start_time = start_timer();

    switch (operation) {
        case 1:
            matrixAddKernel << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b);
            break;
        case 2:
            matrixSubtractKernel << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b);
            break;
        case 3:
            matrixMultiplyKernel << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b);
            break;
        case 4:
            matrixDivideKernel << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b);
            break;

    }
    kernel_time += stop_timer(kernel_start_time, "Kernel execution time:");
    *kernel_runtime = kernel_time;
    //cout<<"Kernel time: "<<kernel_time<<endl;

    //copies result to host so we can use it.
    cudaStatus = hipMemcpy(c, dev_c, sizeof(double) * N, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 3");
        goto Error;
    }

    Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;


}
/*
c is output, a and b are input arrays to perform operation, arrSize is size of array, operation is operation type
*/
void CPUMatrixOperation(double* c, const double* a, const double* b, unsigned int arrSize, int operation){
    switch (operation) {
        case 1:
            CPUMatrixAdd(c, a, b, arrSize);
            break;
        case 2:
            CPUMatrixSubtract(c, a, b, arrSize);
            break;
        case 3:
            CPUMatrixMultiply(c, a, b, arrSize);
            break;  
        case 4:
            CPUMatrixDivide(c, a, b, arrSize);
            break;

    }

}

// Returns the current time in microseconds
long long start_timer() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec * 1000000 + tv.tv_usec;
}

// converts a long long ns value to float seconds
float usToSec(long long time) {
    return ((float)time)/(1000000);
}

// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, const char *name) {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
    float elapsed = usToSec(end_time - start_time);
    printf("%s: %.5f sec\n", name, elapsed);
    return end_time - start_time;
}

