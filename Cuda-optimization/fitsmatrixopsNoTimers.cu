#include "hip/hip_runtime.h"
//Author: Adriel Kim
//6-27-2020
//Updated 7-5-2020
//Updated 7-7-2020
    //Timing with CUDA events to measure PCIe data throughput
/*
Desc: Basic 2D matrix operations - element-wise addition, subtraction, multiplication, and division.

To do:
- Use vector instead of array?
- Error handling for cuda events using a wrapper function
- Be able to test for varying sizes of images. (For now we manually define with constant N)
- Add timer to compare CPU and GPU implementations
- Double check if all memory is freed
- Optimize by eliminating redundant calculations
- Test code on department servers
*/

#include "hip/hip_runtime.h"
#include ""
#include <sys/time.h>
#include <iostream>
#include <fstream>
#include <cstring>
#include <string>
#include <stdlib.h>
#include <stdio.h>

//define imin(a,b)  (a<b?a:b)//example of ternary operator in c++
//4176,2048
#define R 4176
#define C 2048
#define N (R*C)//# of elements in matrices
const int threadsPerBlock = 1024;//threads in a block. A chunk that shares the same shared memory.
const int blocksPerGrid = 8352;//imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);//this will be our output array size for sumKernel.

using namespace std;

hipError_t matrixOperation(double* c, const double* a, const double* b, 
    unsigned int arrSize, int operation, float* kernel_runtime,
     float* GPU_transfer_time, float* cuda_htod_elapsed_time, 
     float* cuda_kernel_elapsed_time, float* cuda_dtoh_elapsed_time,float* cuda_total_time);
void CPUMatrixOperation(double* c, const double* a, const double* b, unsigned int arrSize, int operation);
long long start_timer();
long long stop_timer(long long start_time, const char *name);

//any advantages with mapping directly to strucutre of matrix? We're just representing 2D matrix using 1D array...
//it would be difficult to do the above since we want the operations to occur over abitrarily large matrices
//this can definitely be optimzied by elminating redundant calculations
__global__ void matrixAddKernel(double *c, const double *a, const double *b) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        //adds total number of running threads to tid, the current index.
        tid += blockDim.x * gridDim.x;
    }
}
__global__ void matrixSubtractKernel(double* c, const double* a, const double* b) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] - b[tid];
        //adds total number of running threads to tid, the current index.
        tid += blockDim.x * gridDim.x;
    }
}
__global__ void matrixMultiplyKernel(double* c, const double* a, const double* b) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

__global__ void matrixDivideKernel(double* c, const double* a, const double* b) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = (a[tid]/b[tid]);
        tid += blockDim.x * gridDim.x;
    }
}
//---------------------------------------------------------------------------------
void CPUMatrixAdd(double* c, const double* a, const double* b, unsigned int arrSize){
    for(int i = 0; i < arrSize; i++){
        c[i] = a[i] + b[i];
    }
}
void CPUMatrixSubtract(double* c, const double* a, const double* b, unsigned int arrSize){
    for(int i = 0; i < arrSize; i++){
        c[i] = a[i] - b[i];
    }
}

void CPUMatrixMultiply(double* c, const double* a, const double* b, unsigned int arrSize){
    for(int i = 0; i < arrSize; i++){
        c[i] = a[i] * b[i];
    }
}

void CPUMatrixDivide(double* c, const double* a, const double* b, unsigned int arrSize){
    for(int i = 0; i < arrSize; i++){
        c[i] = a[i] / b[i];
    }
}
//----------------------------------------------------------------------------------
void printMatrix(double* arr) {
    for (int i = 0;i < R; i++) {
        for (int k = 0;k < C; k++) {
            cout << (arr[k + i * R])<<" ";
        }
        cout << endl;
    }
}

void getFileSize(string fileName){
    ofstream binFile;
    ifstream file;
    string line;
    binFile.open("tempBinFile.bin");//I know it's super hacky and gross.
    file.open("./FitTextFiles/"+fileName);
    if(file.is_open()){
        while(getline(file, line)){
            binFile << line;
        }
        file.close();
    }
    streampos begin, end;
    ifstream tempFile ("tempBinFile.bin", ios::binary);
    begin = tempFile.tellg();
    tempFile.seekg(0, ios::end);
    end = tempFile.tellg();
    tempFile.close();
    cout<<"size is: "<< (end-begin)<<" bytes.\n"<<endl;
    remove("tempBinFile.bin");
}

void fillWithFitImage(string fileName, double* arr){
    string line;
    ifstream fitsFile;
    fitsFile.open("./FitTextFiles/"+fileName);
    int ind = 0;
    if(fitsFile.is_open()){
        getline(fitsFile, line);//gets rid of initial part, which is just shape
    }

    if(fitsFile.is_open()){
        while(getline(fitsFile, line) && ind < N){
            char cstr[line.size()+1];
            strcpy(cstr, line.c_str());
            double num = atof(cstr);
            //add num to arr
            arr[ind] = num;
            ind++;
        }
        fitsFile.close();
    }
    else{
        cout<<"Error opening file"<<endl;
    }

}

void fillWithRandomNumbers(double* arr, int arrSize){
    for(int i = 0; i<arrSize;i++){
        arr[i] = rand() % 100+1;
    }

}

void writeImageToText(double *img, string name, int arrSize){
    ofstream file;
    file.open(name);
    for(int i = 0; i<arrSize;i++){
        file << img[i] << "\n";
    }
    file.close();
}

void writeResultToText(string fileName, double result){
    ofstream file;
    file.open(fileName, ios_base::app | ios_base::out);
    file << result << "\n";
    file.close();
}

bool checkEquality(double* arr1, double* arr2, int arrSize){
    for(int i = 0;i < arrSize;i++){
        if (arr1[i]!=arr2[i]){
            return false;
        }
    }
    return true;

}

double getArraySize(int arrSize){
    return arrSize*sizeof(double);
}

int main()
{
    //CUDA Timing variables
    float* cuda_htod = (float*)malloc(sizeof(float));
    float* cuda_kernel_time = (float*)malloc(sizeof(float));
    float* cuda_dtoh = (float*)malloc(sizeof(float));
    float* cuda_total_time = (float*)malloc(sizeof(float));

    //CPU timing variables
    float* GPU_kernel_time = (float*)malloc(sizeof(float));
    float* GPU_transfer_time = (float*)malloc(sizeof(float));

    //Must allocate host memory first before calling kernel.
    double* outputs = (double*)malloc(N * sizeof(double));
    double* doubleMatrix = (double*)malloc(N * sizeof(double));
    double* doubleMatrix2 = (double*)malloc(N * sizeof(double));
    double* CPUoutputs = (double*)malloc(N * sizeof(double));

    int operation = 0;
    
    cout << "Enter which operation (1 = add, 2 = subtract, 3 = multiply, 4 = divide)" << endl;
    cin >> operation;
    //populated 2D array with data
    cout<<"Populating image data"<<endl;
    //fillWithFitImage("imgraw1.txt", doubleMatrix);
    //fillWithFitImage("img1.txt", doubleMatrix2);
    fillWithRandomNumbers(doubleMatrix, N);
    fillWithRandomNumbers(doubleMatrix2, N);

    double arr1Size = getArraySize(N);
    double arr2Size = getArraySize(N);
    double outArrSize = getArraySize(N);

    cout<<"Size of raw image (bytes): " << arr1Size<<endl;
    cout<<"Size of bias image (bytes): "<<arr2Size<<endl;
    //getFileSize("imgraw1.txt");
    //getFileSize("img1.txt");

    cout<<"GPU Start!\n"<<endl;


    hipError_t cudaStatus = matrixOperation(outputs, doubleMatrix, doubleMatrix2, 
        N, operation,GPU_kernel_time, GPU_transfer_time, cuda_htod, 
        cuda_kernel_time, cuda_dtoh, cuda_total_time);




    cout << "CPU Start!\n" << endl;


    CPUMatrixOperation(CPUoutputs, doubleMatrix, doubleMatrix2, N, operation);


    cout << "CPU DONE!" << endl;

    //printMatrix(CPUoutputs);
    bool equal = checkEquality(outputs, CPUoutputs, N);
    if(equal == true)
        cout<<"CPU and GPU outputs are equal"<<endl;
    else
        cout<<"CPU and GPU outputs are NOT equal"<<endl;

    //writeImageToText(outputs,"gpuFit.txt", N);

    free(outputs);
    free(doubleMatrix);
    free(doubleMatrix2);
    free(CPUoutputs);
    free(GPU_kernel_time);
    free(cuda_dtoh);
    free(cuda_htod);
    free(cuda_kernel_time);
    free(cuda_total_time);

    return 0;
}
hipError_t matrixOperation(double* c, const double* a, const double* b, 
    unsigned int arrSize, int operation, float* kernel_runtime, 
    float* GPU_transfer_time, float* cuda_htod_elapsed_time, 
    float* cuda_kernel_elapsed_time, float* cuda_dtoh_elapsed_time, float* cuda_total_time) {


    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    float kernel_time = 0;
    float transfer_time = 0;
    hipError_t cudaStatus;


    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    //Allocate GPU buffers for three vectors (two input, one output)
    cudaStatus = hipMalloc((void**)&dev_c, N * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 1");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, N * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 2");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, N * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 3");
        goto Error;
    }

    //Copy input vectors from host memory to GPU buffers
    cudaStatus = hipMemcpyAsync(dev_a, a, sizeof(double) * N, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 1");
        goto Error;
    }

    cudaStatus = hipMemcpyAsync(dev_b, b, sizeof(double) * N, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 2");
        goto Error;
    }

    switch (operation) {
        case 1:
            matrixAddKernel << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b);
            break;
        case 2:
            matrixSubtractKernel << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b);
            break;
        case 3:
            matrixMultiplyKernel << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b);
            break;
        case 4:
            matrixDivideKernel << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b);
            break;

    }

    //copies result to host so we can use it.
    cudaStatus = hipMemcpy(c, dev_c, sizeof(double) * N, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 3");
        goto Error;
    }



    Error:
    cout<<"Cuda memory freed"<<endl;
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
 
    
}
/*
c is output, a and b are input arrays to perform operation, arrSize is size of array, operation is operation type
*/
void CPUMatrixOperation(double* c, const double* a, const double* b, unsigned int arrSize, int operation){
    switch (operation) {
        case 1:
            CPUMatrixAdd(c, a, b, arrSize);
            break;
        case 2:
            CPUMatrixSubtract(c, a, b, arrSize);
            break;
        case 3:
            CPUMatrixMultiply(c, a, b, arrSize);
            break;  
        case 4:
            CPUMatrixDivide(c, a, b, arrSize);
            break;

    }

}

// Returns the current time in microseconds
long long start_timer() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec * 1000000 + tv.tv_usec;
}

// converts a long long ns value to float seconds
float usToSec(long long time) {
    return ((float)time)/(1000000);
}

// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, const char *name) {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
    float elapsed = usToSec(end_time - start_time);
    printf("%s: %.5f sec\n", name, elapsed);
    return end_time - start_time;
}

